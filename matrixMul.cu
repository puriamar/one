#include "hip/hip_runtime.h"
//This program computes Matrix Multiplication on the GPU using CUDA

#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

__global__ matrixMul(int *a, int *b, int* c, int N){

    //Calculate global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //boundary check for matrix
    if(row < N && col < N){
        int tmp = 0;
        for(int i = 0; i < N; i++)
        {
            tmp += a[row * N + i] * b[i * N + col];
        }   
    }

    //Write Back the result
    c[row * N + col] = tmp;


}

//Initialize matrix of size N * N
void init_matrix(int *m, int N)
{
    for(int i = 0; i < N; i++){
        m[i] = rand() % 100; 
    }
}

//CPU calculation
void cpu(int *a, int *b, int *c, int N)
{
    int tmp;
    //each row
    for(int i = 0; i < N; i++)
    {
        //each col
        for (int j = 0; j < N; j++)
        {
            //each row-col pair
            tmp = 0;
            for (int k = 0; k < N; k++)
            {
                tmp += a[i * N + k] * b[k * N + j];
            }

            //check each result
            assert(tmp == c[i * N + j]);
        }
    }

}

int main(){
    //matrix dimensin, square matrix 2 ^ 10 * 2 ^ 10
    int N = 1 << 10;

    size_t  bytes = N * N * sizeof(int);

    //Allocate memory for matrices
    int *a, int *b, int *c;
    
    hipMallocManaged(&a, bytes); //input matrix of N * N size
    hipMallocManaged(&b, bytes); // input matrix of N * N size
    hipMallocManaged(&c, bytes); // output matrix of N * N size

    //Initialize our matrices
    init_matrix(a, N);
    init_matrix(b, N);

    // Set our block and Grid dimension

    int threads = 16;
    int blocks = (N + threads - 1) / threads;

    //set up kernel launch parameters
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    //launch our kernel
    matrixMul<<<BLOCKS, THREADS>>>(a, b, c, N);

    hipDeviceSynchronize();

    //verify result
    cpu(a, b, c, N);

    cout << "Program completed successfully" << endl;
    return 0;
}